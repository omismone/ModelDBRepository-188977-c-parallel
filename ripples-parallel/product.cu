#include "omislib.cuh"

void productMatMat(struct matrix* c, struct matrix* a, struct matrix* b)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;

    hipblasHandle_t handle;

    hipSetDevice(0);

    hipblasCreate(&handle);

    hipMalloc((void**)&dev_a, a->size[0] * a->size[1] * sizeof(double));
    hipMalloc((void**)&dev_b, b->size[0] * b->size[1] * sizeof(double));
    hipMalloc((void**)&dev_c, c->size[0] * c->size[1] * sizeof(double));

    hipMemcpy(dev_a, a->val, a->size[0] * a->size[1] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b->val, b->size[0] * b->size[1] * sizeof(double), hipMemcpyHostToDevice);

    const double alpha = 1.0;
    const double beta = 0.0;
    int m = b->size[1];
    int n = a->size[0];
    int k = b->size[0];
    hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n, dev_a, k, &beta, dev_c, n);

    hipMemcpy(c->val, dev_c, c->size[0] * c->size[1] * sizeof(double), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}